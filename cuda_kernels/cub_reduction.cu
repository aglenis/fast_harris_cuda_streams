#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
  
     
      // Declare and initialize device pointers for input and output
      int *d_reduce_input, *d_aggregate;
      int num_items = 
     
     
      // Determine temporary device storage requirements for reduction
      void *d_temp_storage = NULL;
      size_t temp_storage_bytes = 0;
      hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_reduce_input, d_aggregate, num_items, hipcub::Max());
 
  // Allocate temporary storage for reduction
  hipMalloc(&d_temp_storage, temp_storage_bytes);
 
    // Run reduction (max)
     hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_reduce_input, d_aggregate, num_items, hipcub::Max(),stream,stream_synchronous);


